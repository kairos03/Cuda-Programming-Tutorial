
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c){
	*c = *a + *b;
}

int main(void) {
	int a, b, c;		// host copies of a, b, c
	int *d_a, *d_b, *d_c;	// device copies of a, b, c
	int size = sizeof(int);
	
	// device space allocation
	hipMalloc((void **) &d_a, size);
	hipMalloc((void **) &d_b, size);
	hipMalloc((void **) &d_c, size);

	// set up inputs
	a = 2;
	b = 7;
	
	// copy inputs
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

	// excute add
	add<<<1,1>>>(d_a, d_b, d_c);

	// copy result
	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

	// print result
	printf("%d\n", c);

	// Cleanup
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}
